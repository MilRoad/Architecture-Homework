#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "stdio.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <ctime>

using namespace std;

hipEvent_t start, end;
float Time = 0;

__global__ void sum_N_1(int *res, int *mas1, int *mas2)
{
	int i = blockIdx.x;
	res[i] = mas1[i] + mas2[i];
}

__global__ void sum_1_N(int *res, int *mas1, int *mas2)
{
	int i = threadIdx.x;
	res[i] = mas1[i] + mas2[i];
}

__global__ void sum(int *res, int *mas1, int *mas2)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	res[i] = mas1[i] + mas2[i];
}

void init(int *mas, int size);

void start_time();

void stop_time();

int main()
{
	int N = 100000, size = 100000;

	int *mas1 = new int[size];
	init(mas1, size);
	int *mas2 = new int[size];
	init(mas2, size);
	int *res = new int[size];

	int *cudaMas1;
	int *cudaMas2;
	int *cudaRes;

	hipMalloc((void**)&cudaMas1, sizeof(int) * size);
	hipMalloc((void**)&cudaMas2, sizeof(int) * size);
	hipMalloc((void**)&cudaRes,  sizeof(int) * size);

	hipMemcpy(cudaMas1, mas1, sizeof(int) * size, hipMemcpyHostToDevice);
	hipMemcpy(cudaMas2, mas2, sizeof(int) * size, hipMemcpyHostToDevice);
	hipMemcpy(cudaRes,  res,  sizeof(int) * size, hipMemcpyHostToDevice);

	hipEventCreate(&start);
	hipEventCreate(&end);

	cout << "Сравним время работы в разных режимах";
	cout << "<<< 1,1 >>>" << endl;
	start_time();
	for (int i = 0; i < size; i++)
		sum<<<1,1>>>(&cudaRes[i], &cudaMas1[i], &cudaMas2[i]);
	hipDeviceSynchronize();
	hipMemcpy(res, cudaRes, sizeof(int) * size, hipMemcpyDeviceToHost);
	stop_time();
	cout << "<<< 1, N >>>" << endl;
	start_time();
	sum_1_N<<<1,N>>>(cudaRes, cudaMas1, cudaMas2);
	hipDeviceSynchronize();
	hipMemcpy(res, cudaRes, sizeof(int) * size, hipMemcpyDeviceToHost);
	stop_time();
	cout << "<<< N,1 >>>" << endl;
	start_time();
	sum_N_1<<<N,1>>>(cudaRes, cudaMas1, cudaMas2);
	hipDeviceSynchronize();
	hipMemcpy(res, cudaRes, sizeof(int) * size, hipMemcpyDeviceToHost);
	stop_time();
	cout << "<<< (Num_elements + (Num_threads - 1))/Num_threads , Num_threads >>>" << endl;
	start_time();
	sum<<<(N + 31) / 32, 32>>>(cudaRes, cudaMas1, cudaMas2);
	hipDeviceSynchronize();
	hipMemcpy(res, cudaRes, sizeof(int) * size, hipMemcpyDeviceToHost);
	stop_time();
	hipFree(cudaMas1);
	hipFree(cudaMas2);
	hipFree(cudaRes);

	return 0;
}


void init(int *mas, int size)
{
	srand( time(0));
	for (int i = 0; i < size; i++)
	{
		mas[i] = rand() % 1000;
	}
} 

void start_time()
{
	hipEventRecord(start);
}

void stop_time()
{
	hipEventRecord(end);
	hipEventSynchronize(end);	
	hipEventElapsedTime(&Time, start, end);
	cout << "Время работы \n" << Time << endl;
}
