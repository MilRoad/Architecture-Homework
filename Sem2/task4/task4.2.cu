#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "stdio.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""

using namespace std;

const int N = 8;
const int block = 4;
hipEvent_t start, end;
float Time = 0;

__global__ void MultMatrCuda(int *res, int *mas1, int *mas2)
{
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int aBegin = N * block * by;
	int aEnd = aBegin + N - 1;
	int aStep = block;

	int bBegin = block * bx;
	int bStep = block * N;

	int sum = 0;

	for (int a = aBegin, b = bBegin; a < aEnd; a += aStep, b += bStep)
	{
		__shared__ int as[block][block];
		__shared__ int bs[block][block];

		as[ty][tx] = mas1[a + N * ty + tx];
		bs[ty][tx] = mas2[b + N * ty + tx];

		__syncthreads();

		for (int k = 0; k < block; k++)
		{
			sum += as[ty][k] * bs[k][tx];

			__syncthreads();
		}

		int c = N * block * by + block * bx;

		res[c + N * ty + tx] = sum;
	}

}

void MultMatr(int *res, int *mas1, int *mas2)
{
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++) {
			for (int k = 0; k < N; k++)
				res[i*N + j] += mas1[i*N + k] * mas2[k*N + j];
		}
	}
}

void PrintMas(int *mas)
{
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++)
			cout << mas[i*N + j] << " ";
		cout << endl;
	}
	cout << endl;
}

void StartTime()
{
	hipEventRecord(start);
}

void StopTime()
{
	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&Time, start, end);
	cout << "Время работы " << Time << endl;
}

int main()
{
	int mas1[N * N] = { 1,1,1,1,1,1,1,1,
						2,2,2,2,2,2,2,2,
						3,3,3,3,3,3,3,3,
						4,4,4,4,4,4,4,4,
						5,5,5,5,5,5,5,5,
						6,6,6,6,6,6,6,6,
						7,7,7,7,7,7,7,7,
						8,8,8,8,8,8,8,8 };
	int mas2[N * N] = { 1,1,1,1,1,1,1,1,
						1,1,1,1,1,1,1,1,
						1,1,1,1,1,1,1,1,
						1,1,1,1,1,1,1,1,
						1,1,1,1,1,1,1,1,
						1,1,1,1,1,1,1,1,
						1,1,1,1,1,1,1,1,
						1,1,1,1,1,1,1,1 };

	int *res = new int[N * N];

	int *cudaMas1;
	int *cudaMas2;
	int *cudaRes;

	hipMalloc((void**)&cudaMas1, sizeof(int) * N * N);
	hipMalloc((void**)&cudaMas2, sizeof(int) * N * N);
	hipMalloc((void**)&cudaRes,  sizeof(int) * N * N);

	hipMemcpy(cudaMas1, mas1, sizeof(int) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(cudaMas2, mas2, sizeof(int) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(cudaRes,  res,  sizeof(int) * N * N, hipMemcpyHostToDevice);

	hipEventCreate(&start);
	hipEventCreate(&end);

	PrintMas(mas1);
	PrintMas(mas2);

	StartTime();
	MultMatr(res, mas1, mas2);
	StopTime();

	cout << "Произведение "<< endl;
	PrintMas(res);

	dim3 threads(block, block);
	dim3 blocks(N / threads.x, N / threads.y);

	StartTime();
	MultMatrCuda<<< blocks, threads >>>(cudaRes, cudaMas1, cudaMas2);
	StopTime();
	hipMemcpy(res, cudaRes, sizeof(int) * N * N, hipMemcpyDeviceToHost);

	cout << "Произведение" << endl;
	PrintMas(res);
 	
	hipFree(cudaMas1);
	hipFree(cudaMas2);
	hipFree(cudaRes);

	return 0;
}
