#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "stdio.h"
#include <iostream>



using namespace std;

__global__ void SumMatrCuda(int *res, int *mas1, int *mas2, int N, int M)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int tx = i * N + j;
	res[tx] = mas1[tx] + mas2[tx];
}

void SumMatr(int *res, int *mas1, int *mas2, int N, int M)
{
	for (int i = 0; i < N; i++){
		for (int j = 0; j < M; j++)
			res[i*N + j] = mas1[i*N + j] + mas2[i*N + j];
	}
}

void PrintMas(int *mas, int N, int M)
{
	for (int i = 0; i < N; i++){
		for (int j = 0; j < M; j++)
			cout << mas[i*N + j] << " ";
		cout << endl;
	}
	cout << endl;
}

int main()
{
	const int N = 4, M = 4;
	int dimN = 2, dimM = 2;

	int mas1[N * M] = {1,2,3,4,
					5,6,7,8,
					9,10,11,12,
					13,14,15,16};
	int mas2[N * M] = {16,15,14,13,
						12,11,10,9,
						8,7,6,5,
						4,3,2,1};
	int *res = new int[N * M];

	int *cudaMas1;
	int *cudaMas2;
	int *cudaRes;

	float time = 0;

	hipMalloc((void**)&cudaMas1, sizeof(int) * N * M);
	hipMalloc((void**)&cudaMas2, sizeof(int) * N * M);
	hipMalloc((void**)&cudaRes,  sizeof(int) * N * M);

	hipMemcpy(cudaMas1, mas1, sizeof(int) * N * M, hipMemcpyHostToDevice);
	hipMemcpy(cudaMas2, mas2, sizeof(int) * N * M, hipMemcpyHostToDevice);
	hipMemcpy(cudaRes,  res,  sizeof(int) * N * M, hipMemcpyHostToDevice);

	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);

	PrintMas(mas1, N, M);
	PrintMas(mas2, N, M);

	hipEventRecord(start);
	SumMatr(res, mas1, mas2, N, M);
	hipEventRecord(end);
	hipEventSynchronize(end);	

	cout << "Cумма "<< endl;
	PrintMas(res, N, M);

	hipEventElapsedTime(&time, start, end);
	cout << "Время работы в обычном режиме " << time << endl;

	dim3 blocks(N / dimN, M / dimM);
	dim3 threads(dimN, dimM);

	hipEventRecord(start);
	SumMatrCuda<<< blocks, threads >>>(cudaRes, cudaMas1, cudaMas2, N, M);
	hipDeviceSynchronize();
	hipEventRecord(end);

	hipMemcpy(res, cudaRes, sizeof(int) * N * M, hipMemcpyDeviceToHost);

	cout << "Сумма" << endl;
	PrintMas(res, N, M);
 	hipEventElapsedTime(&time, start, end);
   	cout << "Время работы в параллельном режиме " << time << endl;

	hipFree(cudaMas1);
	hipFree(cudaMas2);
	hipFree(cudaRes);

	return 0;
}
